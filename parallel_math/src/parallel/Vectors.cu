#include "hip/hip_runtime.h"
//
// Created by Santiago Hurtado on 18.10.19.
//
#include "parallel/Vectors.h"
#include <stdio.h>

__global__ void kernel(unsigned int* a,unsigned int* b,unsigned int* result, int size) {
    auto count = blockIdx.x * blockDim.x + threadIdx.x;
    if(count<size){
        result[count]=a[count]+b[count];
    }
}

namespace parallel{
    void Vectors::sum_size(size_t size) {
        this->result.clear();
        auto mem_size = sizeof(unsigned int)*size;
        // Host memory management
        auto h_a = new unsigned int[size];
        auto h_b = new unsigned int[size];
        auto result = new unsigned int[size];
        for(auto i =0;i<size;++i){
            h_a[i]=i;
            h_b[i]=i*i;
        }
        // Device memory management
        unsigned int *d_a;
        unsigned int *d_b;
        unsigned int *d_result;

        hipMalloc(&d_a,mem_size);
        hipMalloc(&d_b,mem_size);
        hipMalloc(&d_result,mem_size);

        hipMemcpy(d_a, h_a, mem_size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, mem_size, hipMemcpyHostToDevice);

        kernel<<<(size+1023)/1024,1024>>>(d_a,d_b,d_result,size);
        hipMemcpy(result,d_result,mem_size,hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
        for(auto i=0;i<size;++i){
            this->result.push_back(result[i]);
        }
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_result);

        delete [] h_a;
        delete [] h_b;
        delete [] result;
    }

    void Vectors::sum(const std::vector<unsigned int> &a, const std::vector<unsigned int> &b) {
        this->result.clear();
        if(a.empty()||b.empty()||a.size()!=b.size()){
            return;
        }
        auto size = a.size();
        auto mem_size = sizeof(unsigned int)*size;

        // Host memory management
        auto a_a = std::make_unique<unsigned int[]>(size);
        auto a_b = std::make_unique<unsigned int[]>(size);
        auto result = std::make_unique<unsigned int[]>(size);
        std::copy(a.begin(), a.end(), a_a.get());
        std::copy(b.begin(), b.end(), a_b.get());

        // Device memory management
        unsigned int *d_a;
        unsigned int *d_b;
        unsigned int *d_result;

        hipMalloc(&d_a,mem_size);
        hipMalloc(&d_b,mem_size);
        hipMalloc(&d_result,mem_size);

        hipMemcpy(d_a,a_a.get(),mem_size,hipMemcpyHostToDevice);
        hipMemcpy(d_b,a_b.get(),mem_size,hipMemcpyHostToDevice);

        kernel<<<(size+1023)/1024,1024>>>(d_a,d_b,d_result,size);

        hipMemcpy(result.get(),d_result,mem_size,hipMemcpyDeviceToHost);

        for(auto i=0;i<size;++i){
            this->result.push_back(result[i]);
        }

        hipDeviceSynchronize();
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_result);
    }
}
